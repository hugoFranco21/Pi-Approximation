#include "hip/hip_runtime.h"
/*----------------------------------------------------------------
* Multiprocesadores: Proyecto final
* Fecha: 21-Nov-2021
* Autor: A01654856 Hugo David Franco Ávila
* Descripción: Este código implementa la serie de Nilikantha para
    obtener una aproximación suficientemente precisa de Pi en el lenguaje CUDA C. A medida
    que incrementan los términos de la serie, la aproximación es más precisa.
    El algoritmo está paralelizado, utilizando la tecnología de CUDA de NVIDIA.
    Hago uso del archivo utils.h generado por el Prof. Pedro Pérez
* Comando para compilar en Linux: nvcc nilikantha.cu
* Comando para correr en Linux: ./a.out
* NOTA: De especificar otro nombre del ejecutable al compilar utilizar ./nombreDelEjecutable.out
*--------------------------------------------------------------*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000 //1e9
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

/**
 * @brief This function will be run on the device (NVIDIA GPU), that is indicated by the
 * __global__ keyword. It will calculate the Nikilantha series for n terms, and save every 
 * partial sum for every block in the cache, which will be further reduced to be saved in 
 * the result array passed as parameter.
 * 
 * @param size int
 * @return double 
 */
__global__ void nilik(double *result) {
	/**
	 * The array declared below, will be shared across all blocks, and will
	 * save the partial sum of the Nilikantha terms calculated in the block
	 */
	__shared__ double cache[THREADS];

	/**
	 * The tid is a linealization of the memory in the GPU, and will be used
	 * for the calculation as the nth term
	 */
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

	double acum = 0.0, sign, di, denominator, term;
	/**
	 * Special case for the first element in the series
	 */
    if(tid == 0){
        acum += 3.0;
        tid += blockDim.x * gridDim.x;
    }

	/**
	 * The while loop below obtains every Nilikantha term for every threadId index in
	 * every block.
	 */
	while (tid < SIZE) {
		sign = tid % 2 == 0 ? -1.0 : 1.0;
        di = tid*2.0;
        denominator = (di*(di+1)*(di+2));
        term = (4.0*sign)/denominator;
        acum += term;
		tid += blockDim.x * gridDim.x;
	}

	/**
	 * Partial sum is saved in the cache
	 */
	cache[cacheIndex] = acum;

	/**
	 * The instruction below performs a block level syncronization barrier, this 
	 * means it will be called when every thread reaches this line in their execution pipeline
	 */
	__syncthreads();

	/**
	 * The code below, performs a reduction by adding the contents of the element in a power of 2
	 * distance to the element at the cacheIndex, which if recalled is the same as the threadId,
	 * it will stop after it reaches the original index. 
	 */
	int i = blockDim.x / 2;
	while (i > 0) {
		/**
		 * The if block belows prevents accidentaly accesing a non-valid index in the device.
		 */
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		/**
		 * The line belows blocks all threads from advancing until they reach this line in their
		 * execution flow.
		 */
		__syncthreads();
		i /= 2;
	}

	/**
	 * After the contents in the block are summed up in each index of the cache, 
	 * it wil be added to the result array at the position indicated by its blockId
	 */
	if (cacheIndex == 0) {
		result[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
    if(argc >= 2){
        printf("Error: No arguments are allowed\n");
        return -1;
    }
	int i;
	double *results, *d_r;
	double ms;

	/**
	 * Cache allocation in the host memory
	 */
	results = (double*) malloc( BLOCKS * sizeof(double) );

	/**
	 * Cache allocation in the device memory
	 */
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(double) );

	printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();
		/**
		 * This command calls the code defined in the function nilik and specifies the number of blocks
		 * and threads to be used for the calculation, the parameter is the cache array 
		 * allocated in the device.
		 */
		nilik<<<BLOCKS, THREADS>>> (d_r);

		ms += stop_timer();
	}

	/**
	 * Results from the device are copied onto the host 
	 */
	hipMemcpy(results, d_r, BLOCKS * sizeof(double), hipMemcpyDeviceToHost);

	/**
	 * The block below performs a reduction in the results array, getting the approximation of Pi.
	 * By using the parallel reduction earlier in the nilik function, it allowed us to only 
	 * need to perform n-block amount of operations to get the result, instead of adding the 
	 * result of every thread.
	 */
	double acum = 0;
	for (i = 0; i < BLOCKS; i++) {
		acum += results[i];
	}

	printf("Pi is approx: %.15lf \n", acum);
    printf("Average time elapsed: %.3lf ms\n", (ms / N));

	/**
	 * De-allocation of device memory
	 */
	hipFree(d_r);

	/**
	 * De-allocation of host memory
	 */
	free(results);
	return 0;
}
